//Desription: Image processing algorithm for "pyramid generation". See "Pyramidal Implementation of the Lucas Kanade Feature Tracker Description of the algorithm" for a detailed description
//It uses a “tiled convolution” structure, where each block is responsible for generating a corresponding “tile” on the pyramid image.


#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>
#include <math.h>

//Time stamp function in seconds
double getTimeStamp() {
   struct timeval tv;
   gettimeofday(&tv, NULL);
   return (double) tv.tv_usec/1000000 + tv.tv_sec;
}

//When generating multiple levels of pyramids, i corresponds to the pyramid level the current kernel call is generating
__global__ void generatePyramid (float *pyramids, int i, int origImgWidth, int origImgHeight) {
  int ix = threadIdx.x + blockIdx.x*blockDim.x;
  int iy = threadIdx.y + blockIdx.y*blockDim.y ;

  int origImgSize = origImgWidth*origImgHeight;
  
  int imgOffset = int(origImgSize * (1-pow(0.25, i-1))/(1-0.25));
  int pyramidOffset = int(origImgSize * (1-pow(0.25, i))/(1-0.25));

  int imgHeight = origImgHeight >> (i-1);
  int imgWidth = origImgWidth >> (i-1);
  
  int pyrmHeight = imgHeight >> 1;
  int pyrmWidth = imgWidth >> 1;
  
  int idx = iy* pyrmWidth + ix ;
  
  extern __shared__ float sImg [];

  //Move data block uses to shared memory for faster reads
  //center
  sImg[threadIdx.x*2 + 1 + (threadIdx.y*2 + 1)*(blockDim.x*2 + 1)] = pyramids[imgOffset + iy*2*imgWidth + ix*2];
  //center right
  sImg[threadIdx.x*2 + 2 + (threadIdx.y*2 + 1)*(blockDim.x*2 + 1)] = pyramids[imgOffset + iy*2*imgWidth + ix*2 + 1];
  //bottom center
  sImg[threadIdx.x*2 + 1 + (threadIdx.y*2 + 2)*(blockDim.x*2 + 1)] = pyramids[imgOffset + (iy*2 + 1)*imgWidth + ix*2];
  //bottom right
  sImg[threadIdx.x*2 + 2 + (threadIdx.y*2 + 2)*(blockDim.x*2 + 1)] = pyramids[imgOffset + (iy*2 + 1)*imgWidth + ix*2 + 1];
  
  if (threadIdx.y == 0) {
     //top center
     sImg[threadIdx.x*2 + 1 + (threadIdx.y*2)*(blockDim.x*2 + 1)] = blockIdx.y == 0 ? pyramids[imgOffset + iy*2*imgWidth + ix*2]:pyramids[imgOffset + (iy*2-1)*imgWidth + ix*2];
     //top right
     sImg[threadIdx.x*2 + 2 + (threadIdx.y*2)*(blockDim.x*2 + 1)] = blockIdx.y == 0 ? pyramids[imgOffset + iy*2*imgWidth + ix*2 + 1]:pyramids[imgOffset + (iy*2-1)*imgWidth + ix*2 + 1];	  
  }
  if (threadIdx.x == 0) {
     //center left
     sImg[threadIdx.x*2 + (threadIdx.y*2 + 1)*(blockDim.x*2 + 1)] = blockIdx.x == 0 ? pyramids[imgOffset + iy*2*imgWidth + ix*2]:pyramids[imgOffset + iy*2*imgWidth + ix*2 - 1];
     //bottom left
     sImg[threadIdx.x*2 + (threadIdx.y*2 + 2)*(blockDim.x*2 + 1)] = blockIdx.x == 0 ? pyramids[imgOffset + (iy*2 + 1)*imgWidth + ix*2]:pyramids[imgOffset + (iy*2 + 1)*imgWidth + ix*2 - 1];	  
  }
  if (threadIdx.x == 0 && threadIdx.y == 0)
     //top left
     sImg[threadIdx.x*2 + (threadIdx.y*2)*(blockDim.x*2 + 1)] = (blockIdx.x == 0 || blockIdx.y == 0) ? pyramids[imgOffset + iy*2*imgWidth + ix*2]:pyramids[imgOffset + (iy*2-1)*imgWidth + ix*2 - 1];
   
  __syncthreads();
    
  if( (ix<pyrmWidth) && (iy<pyrmHeight) ) {
   
      #ifdef DEBUG
	 int centerX = min(max(2*ix, 0), imgWidth);
         int centerY = min(max(2*iy, 0), imgHeight);

         int left = min(max(2*ix - 1, 0), imgWidth);
         int down = min(max(2*iy - 1, 0), imgHeight);

         int right = min(max(2*ix + 1, 0), imgWidth);
         int up = min(max(2*iy + 1, 0), imgHeight);
	  
         printf("Index: (%d, %d)\n", ix, iy);
      	 printf("Center (%d, %d): %lf\n", centerX, centerY, pyramids[centerX + centerY*imgWidth + imgOffset]);
         printf("Center Left (%d, %d): %lf\n", left, centerY, pyramids[left + centerY*imgWidth + imgOffset]);
         printf("Center Right (%d, %d): %lf\n", right, centerY, pyramids[right + centerY*imgWidth + imgOffset]);	    
         printf("Up Left (%d, %d): %lf\n", left, up, pyramids[left + up*imgWidth + imgOffset]);
         printf("Up Center (%d, %d): %lf\n", centerX, up, pyramids[centerX + up*imgWidth + imgOffset]);	    
         printf("Up Right (%d, %d): %lf\n", right, up, pyramids[right + up*imgWidth + imgOffset]);
         printf("Down Left (%d, %d): %lf\n", left, down, pyramids[left + down*imgWidth + imgOffset]);
         printf("Down Center (%d, %d): %lf\n", centerX, down, pyramids[centerX + down*imgWidth + imgOffset]);	    
         printf("Down Right (%d, %d): %lf\n", right, down, pyramids[right + down*imgWidth + imgOffset]);
      #endif

      float pValue = 0;
      
      pValue += sImg[threadIdx.x*2 + 1 + (threadIdx.y*2 + 1)*(blockDim.x*2 + 1)]/4.0;
      pValue += 1/8*sImg[threadIdx.x*2 + 2 + (threadIdx.y*2 + 1)*(blockDim.x*2 + 1)]/8.0;
      pValue += 1/8*sImg[threadIdx.x*2 + 1 + (threadIdx.y*2 + 2)*(blockDim.x*2 + 1)]/8.0;
      pValue += 1/16*sImg[threadIdx.x*2 + 2 + (threadIdx.y*2 + 2)*(blockDim.x*2 + 1)]/16.0;
      pValue += 1/8*sImg[threadIdx.x*2 + 1 + (threadIdx.y*2)*(blockDim.x*2 + 1)]/8.0;
      pValue += 1/16*sImg[threadIdx.x*2 + 2 + (threadIdx.y*2)*(blockDim.x*2 + 1)]/16.0;
      pValue += 1/8*sImg[threadIdx.x*2 + (threadIdx.y*2 + 1)*(blockDim.x*2 + 1)]/8.0;
      pValue += 1/16*sImg[threadIdx.x*2 + (threadIdx.y*2 + 2)*(blockDim.x*2 + 1)]/16.0;
      pValue += 1/16*sImg[threadIdx.x*2 + (threadIdx.y*2)*(blockDim.x*2 + 1)]/16.0;

      pyramids[idx + pyramidOffset] = pValue;
  }
}
int main() {
   FILE *img = fopen("img.bmp", "rb");

   unsigned char info[54];
   fread(info, sizeof(unsigned char), 54, img); // read the 54-byte header

   //Extract image height and width from header
   int imgWidth = *(int*)&info[18];
   int imgHeight = *(int*)&info[22];
   printf("Size: %d %d\n", imgWidth, imgHeight);
   
   int imgSize = imgWidth * imgHeight;
   unsigned char* data = (unsigned char*)malloc(3*imgSize*sizeof(unsigned char)); // allocate 3 bytes per pixel
   fread(data, sizeof(unsigned char), 3*imgSize, img); // read the rest of the data at once
   fclose(img);

   int pyramidLevels = 3;

   //In terms of #elements
   //Geometric series formula
   int pyramidsSize = int(imgSize * (1-pow(0.25, pyramidLevels+1))/(1-0.25)); 

   float* h_pyramids;
   hipHostAlloc( (void **) &h_pyramids, pyramidsSize*sizeof(float), 0) ;

   //Init data...
   for (int i = 0; i < imgHeight; i++) {
      for (int j = 0; j < imgWidth; j++) {
         h_pyramids[i*imgWidth + j] =  data[((imgHeight - 1 - i)*imgWidth + j)*3];
      }
   }
   
   float* d_pyramids;
   hipMalloc((void**) &d_pyramids, pyramidsSize*sizeof(float));
	     
   double timeStampA = getTimeStamp() ;
      
   hipMemcpy(d_pyramids, h_pyramids, imgSize*sizeof(float), hipMemcpyHostToDevice);
   double timeStampB = getTimeStamp() ;
   
   dim3 block(16, 16);
   
   for (int i = 1; i <= pyramidLevels; i++) {
       int gridX = ceil((imgWidth >> i)/block.x);
       int gridY = ceil((imgWidth >> i)/block.y);
       
       dim3 grid(gridX, gridY);
       generatePyramid<<<grid, block, (2*block.x + 1) * (2*block.y + 1)*sizeof(float)>>> (d_pyramids, i, imgWidth, imgHeight);		 
   }
   
   hipDeviceSynchronize();
   double timeStampC = getTimeStamp() ;
      
   hipMemcpy (h_pyramids, d_pyramids, pyramidsSize*sizeof(float), hipMemcpyDeviceToHost);

   double timeStampD = getTimeStamp();
   
   hipError_t err = hipPeekAtLastError();

   if (err != hipSuccess) {
      printf("Error: %s", hipGetErrorString(err));
      exit(-1);
   }

   hipFree(d_pyramids);
   hipHostFree(h_pyramids);
   hipDeviceReset();

   printf("\n\n\n%.6f\n", timeStampB-timeStampA);
   printf("%.6f\n", timeStampC-timeStampA);
   printf("%.6f\n", timeStampD-timeStampC);
}
